



#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <unistd.h>
#include <stdio.h>
#include <string.h>
#include <omp.h>
#include <iostream>

#define N 900000000 //min:10 max: 900000000 (unsigned int will reach maximum size)

using namespace std;

void warmUpGPU();
__global__ void vectorAdd(unsigned int * A, unsigned int * B, unsigned int * C);

int main(int argc, char *argv[])
{
	
	warmUpGPU();
	
	//change OpenMP settings:
	omp_set_num_threads(1);


	unsigned int * A;
	unsigned int * B;
	unsigned int * C;
	unsigned int * C_CPU;

	A=(unsigned int *)malloc(sizeof(unsigned int)*N);
	B=(unsigned int *)malloc(sizeof(unsigned int)*N);
	C=(unsigned int *)malloc(sizeof(unsigned int)*N);
	C_CPU=(unsigned int *)malloc(sizeof(unsigned int)*N);


	printf("\nSize of A+B+C (GiB): %f",(sizeof(unsigned int)*N*3.0)/(1024.0*1024.0*1024.0));
	

	//init:
	unsigned int i=0;
	for (i=0; i<N; i++){
		A[i]=i;
		B[i]=i;
		C[i]=0;
		C_CPU[i]=0;
	}


	//CPU version:
	double tstartCPU=omp_get_wtime();
	
	for (int i=0; i<N; i++){
		C_CPU[i]=A[i]+B[i];
	}
	double tendCPU=omp_get_wtime();

	printf("\nTime CPU (s): %f",tendCPU - tstartCPU);

	

	double tstart=omp_get_wtime();

	
	//CUDA error code:
	
	hipError_t errCode=hipSuccess;
	
	if(errCode != hipSuccess)
	{
		cout << "\nLast error: " << errCode << endl; 	
	}

	unsigned int * dev_A;
	unsigned int * dev_B;
	unsigned int * dev_C;

	//allocate on the device: A, B, C
	errCode=hipMalloc((unsigned int**)&dev_A, sizeof(unsigned int)*N);	
	if(errCode != hipSuccess) {
	cout << "\nError: A error with code " << errCode << endl; 
	}

	errCode=hipMalloc((unsigned int**)&dev_B, sizeof(unsigned int)*N);	
	if(errCode != hipSuccess) {
	cout << "\nError: B error with code " << errCode << endl; 
	}

	errCode=hipMalloc((unsigned int**)&dev_C, sizeof(unsigned int)*N);	
	if(errCode != hipSuccess) {
	cout << "\nError: C error with code " << errCode << endl; 
	}

	//copy A to device
	errCode=hipMemcpy( dev_A, A, sizeof(unsigned int)*N, hipMemcpyHostToDevice);
	if(errCode != hipSuccess) {
	cout << "\nError: A memcpy error with code " << errCode << endl; 
	}	
	
	//copy B to device
	errCode=hipMemcpy( dev_B, B, sizeof(unsigned int)*N, hipMemcpyHostToDevice);
	if(errCode != hipSuccess) {
	cout << "\nError: A memcpy error with code " << errCode << endl; 
	}

	//copy C to device (initialized to 0)
	errCode=hipMemcpy( dev_C, C, sizeof(unsigned int)*N, hipMemcpyHostToDevice);
	if(errCode != hipSuccess) {
	cout << "\nError: A memcpy error with code " << errCode << endl; 
	}

	//execute kernel
	const unsigned int totalBlocks=ceil(N*1.0/1024.0);
	printf("\ntotal blocks: %d",totalBlocks);
	vectorAdd<<<totalBlocks,1024>>>(dev_A, dev_B, dev_C);

	if(errCode != hipSuccess){
		cout<<"Error afrer kernel launch "<<errCode<<endl;
	}

	//copy data from device to host 
	errCode=hipMemcpy( C, dev_C, sizeof(unsigned int)*N, hipMemcpyDeviceToHost);
	if(errCode != hipSuccess) {
	cout << "\nError: getting C result form GPU error with code " << errCode << endl; 
	}



	hipDeviceSynchronize();

	//testing
	for (int i=N-10; i<N; i++)
	{
		printf("\n%d",C[i]);
	}
	
	
	double tend=omp_get_wtime();
	
	printf("\nTotal time GPU (s): %f",tend-tstart);



	printf("\n");

	return 0;
}

__global__ void vectorAdd(unsigned int * A, unsigned int * B, unsigned int * C) {

unsigned int tid=threadIdx.x+ (blockIdx.x*blockDim.x); 

if (tid>=N){
	return;
}
C[tid]=A[tid]+B[tid];

return;
}

























__global__ void warmup(unsigned int * tmp) {
if (threadIdx.x==0)
*tmp=555;

return;
}



void warmUpGPU(){


printf("\nWarming up GPU for time trialing...\n");	
hipDeviceSynchronize();

return;
}