#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <unistd.h>
#include <stdio.h>
#include <string.h>
#include <math.h>
#include <omp.h>
#include <iostream>

//See values of N in assignment instructions.

#define N 1000
//Do not change the seed, or your answer will not be correct
#define SEED 72

//For GPU implementation
#define BLOCKSIZE 1024

using namespace std;

struct pointData{
double x;
double y;
};

//void warmUpGPU();
void generateDataset(struct pointData * data);
__global__ void calcDistance(struct pointData * pointData, unsigned int * count, double * epsilon);

int main(int argc, char *argv[])
{

        //warmUpGPU();

        //Read epsilon distance from command line
        if (argc!=2)
        {
        printf("\nIncorrect number of input parameters. Please input an epsilon distance.\n");
        return 0;
        }


        char inputEpsilon[20];
        strcpy(inputEpsilon,argv[1]);
        double epsilon=atof(inputEpsilon);


        //generate dataset:
        struct pointData * data;
        data=(struct pointData*)malloc(sizeof(struct pointData)*N);
        printf("\nSize of dataset (MiB): %f",(2.0*sizeof(double)*N*1.0)/(BLOCKSIZE * BLOCKSIZE));
        generateDataset(data);

        omp_set_num_threads(1);

        hipError_t errCode=hipSuccess;

        struct pointData * dev_pointData;
        double dev_epsilon;

        // start time for total time (including memory allocation and copying)
        double tstart=omp_get_wtime();

        // allocate on the device
        errCode=hipMalloc((struct pointData**)&dev_pointData, sizeof(struct pointData)*N);
        if(errCode != hipSuccess) {
        cout << "\nError: A error with code " << errCode << endl;
        }
        // copy  to the device
        errCode=hipMemcpy( dev_pointData, data, sizeof(struct pointData)*N, hipMemcpyHostToDevice);
        if(errCode != hipSuccess) {
        cout << "\nError: A memcpy error with code " << errCode << endl;
        }

        // copy epsilon over
        /*
        errCode=hipMalloc((double)dev_epsilon, sizeof(double));
        if(errCode != hipSuccess) {
        cout << "\nError: B error with code " << errCode << endl;
        }

        errCode=hipMemcpy( &dev_epsilon, epsilon, sizeof(double), hipMemcpyHostToDevice);
        if(errCode != hipSuccess) {
        cout << "\nError: B memcpy error with code " << errCode << endl;
        }
        */

        unsigned int * count;
        unsigned int * dev_count;

        count=(unsigned int *)malloc(sizeof(unsigned int));
        dev_count=(unsigned int *)malloc(sizeof(unsigned int));
        *count = 0;

        //copy count over
        errCode=hipMalloc((unsigned int**)&dev_count, sizeof(unsigned int));
        if(errCode != hipSuccess) {
        cout << "\nError: C hipMalloc error with code " << errCode << endl;
        }
        
        errCode=hipMemcpy( dev_count, count, sizeof(unsigned int), hipMemcpyHostToDevice);
        if(errCode != hipSuccess) {
        cout << "\nError: C memcpy error with code " << errCode << endl;
        }


        //hipDeviceSynchronize()


        // execute the kernel
        // time the kernel
        double tKernelStart=omp_get_wtime();

        const unsigned int totalBlocks=ceil(N*1.0/BLOCKSIZE);
        printf("\ntotal blocks: %d",totalBlocks);

        calcDistance<<<totalBlocks,BLOCKSIZE>>>(dev_pointData, dev_count, epsilon);

        hipDeviceSynchronize();

        double tKernelEnd=omp_get_wtime();


        //copy data from device to host
        errCode=hipMemcpy( count, dev_count, sizeof(unsigned int), hipMemcpyDeviceToHost);
        if(errCode != hipSuccess) {
        cout << "\nError: getting result form GPU error with code " << errCode << endl;
        }

        printf("\nTotal count: %d",*count);

        // get total time
        double tend=omp_get_wtime();

        printf("\nTotal time (s): %f",tend-tstart);
        printf("\nGPU Kernel time (s): %f",tKernelEnd-tKernelStart);

        free(data);
        printf("\n");
        return 0;
}

__global__ void calcDistance(struct pointData * data, unsigned int * count, double epsilon)
{
        //unsigned int tid=threadIdx.x+ (blockIdx.x*blockDim.x);
        unsigned int tid = threadIdx.x + BLOCKSIZE * blockIdx.x;

        if (tid>=N){
        return;
        }

        // attempt at shared memory
        __shared__ double sharedEpsilon_squared;
        sharedEpsilon_squared = epsilon * epsilon;

        __shared__ struct pointData sharedData[BLOCKSIZE];
        // __shared__ struct pointData sharedData2[N]; <-- illegal...?

        //tile data from data to array
        sharedData[tid] = data[tid];
        //sharedData2[tid] = data[tid];

        __syncthreads();

        int i;

        for(i=0; i<N; i++)
        {
                if ( fabs( (sharedData[tid].x - sharedData[i].x ) * (sharedData[tid].x - sharedData[i].x) )
                   + fabs( (sharedData[tid].y - sharedData[i].y ) * (sharedData[tid].y - sharedData[i].y) )
                   <= ( sharedEpsilon_squared ) )
                {
                        atomicAdd(count, int(1));
                }
        }


        return;
}

//Do not modify the dataset generator or you will get the wrong answer
void generateDataset(struct pointData * data)
{

        //seed RNG
        srand(SEED);


        for (unsigned int i=0; i<N; i++){
                data[i].x=1000.0*((double)(rand()) / RAND_MAX);
                data[i].y=1000.0*((double)(rand()) / RAND_MAX);
        }


}
     
